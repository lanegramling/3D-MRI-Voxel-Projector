#include "hip/hip_runtime.h"
#include <iostream>
#include <fstream>
#include <stdio.h>
#include <string>
#include <math.h> //For round()


#include "ImageWriter.h"


// MAX kernel - Computes results of MAX image into device buffer, in
// as well as the working buffer for the SUM computation during traversal.
__global__
void maxImage(unsigned char* d_voxelData, unsigned char* d_maxBuf, float* d_workBuf, float* d_maxWeightedSum, int zDepth)
{
	//Each thread receives a vector that will be reduced to a single pixel and
	// placed in the buffer. Operate on the given thread's work assignment within
	// the voxel buffer using the ID and offset calculation.


	//Calculate the xy offset of the vector to traverse
	int xy_offset = blockIdx.x * blockDim.x + threadIdx.x;

	unsigned char max = d_voxelData[xy_offset];
	float sum = d_voxelData[xy_offset];

	//Perform computations.
	for (int z_offset=1; z_offset < zDepth; z_offset++) {
		int i = z_offset * blockDim.x * gridDim.x + xy_offset;
		sum += (d_voxelData[i] + 1) / zDepth; //Sum, weighted for distance to front.
		if (max < d_voxelData[i]) max = d_voxelData[i];
	}
	atomicMax((int*)d_maxWeightedSum, (int)sum); //Atomically update the max weighted sum if necessary.

	d_maxBuf[xy_offset] = max;
	d_workBuf[xy_offset] = sum;
}

// SUM kernel - Computes results of SUM image into device buffer by
// normalizing the working buffer computed in the MAX Kernel.
__global__
void sumImage(unsigned char* d_voxelData, unsigned char* d_sumBuf, float* d_workBuf, float* d_maxWeightedSum, int zDepth)
{
	int xy_offset = blockIdx.x * blockDim.x + threadIdx.x;

	d_sumBuf[xy_offset] = ((d_workBuf[xy_offset] / *d_maxWeightedSum) * 255.0);
}


// int project(int projectionType, int xSize, int ySize, int zSize)
// {
// 	//Traversal directions are determined based on projection type.
//
//
// }


void writeImage(std::string fName, int xres, int yres, const unsigned char* imageBytes)
{
	unsigned char* row = new unsigned char[3*xres];
	ImageWriter* w = ImageWriter::create(fName,xres,yres);
	int next = 0;
	for (int r=0 ; r<yres ; r++)
	{
		for (int c=0 ; c<3*xres ; c+=3)
		{
			row[c] = row[c+1] = row[c+2] = imageBytes[next++];
		}
		w->addScanLine(row);
	}
	w->closeImageFile();
	delete w;
	delete [] row;
}


//Used for projection. Orients comparison for whether descending or ascending.
bool cmp(int num, int boundary)
{ return (!boundary) ? num > 0 : num < boundary; }

//main
int main(int argc, char* argv[])
{
	//Report versions
	int driverVersion, runtimeVersion;
	hipError_t dv = hipDriverGetVersion(&driverVersion);
	hipError_t rv = hipRuntimeGetVersion(&runtimeVersion);
	std::cout << "Driver version: " << driverVersion << "; Runtime version: " << runtimeVersion << "\n\n";


	//Grab and validate input
	if (argc != 7) {
		std::cout << "\nERROR - Incorrect syntax.\nUsage:\n   ./executive nRows nCols nSheets fileName projectionType outputFileNameBase\n";
		exit(1);
	}
	int nRows = std::atoi(argv[1]);
	int nCols = std::atoi(argv[2]);
	int nSheets = std::atoi(argv[3]);
	char* fileName = argv[4];
	int projectionType = std::atoi(argv[5]);
	char* outputFileNameBase = argv[6];


	//Orient according to projection.
	// array indices are always ordered as x, y, z
	int dir[3];
	int start[3];
	int end[3];
	switch(projectionType) {
		case 1:	//Traverse SHEET (min -> max)
			dir[0] = 1;
			dir[1] = 1;
			dir[2] = 1;
			start[0] = 0;
			start[1] = 0;
			start[2] = 0;
			end[0] = nCols;
			end[1] = nRows;
			end[2] = nSheets;
			break;
		case 2: //Traverse SHEET (max -> min)
			dir[0] = -1;
			dir[1] = 1;
			dir[2] = -1;
			start[0] = nCols;
			start[1] = 0;
			start[2] = nSheets;
			end[0] = 0;
			end[1] = nRows;
			end[2] = 0;
			break;
		case 3: //Traverse   COL (max -> min)
			dir[0] = 1;
			dir[1] = 1;
			dir[2] = -1;
			start[0] = 0;
			start[1] = 0;
			start[2] = nCols;
			end[0] = nSheets;
			end[1] = nRows;
			end[2] = 0;
			break;
		case 4: //Traverse   COL (min -> max)
			dir[0] = -1;
			dir[1] = 1;
			dir[2] = 1;
			start[0] = nSheets;
			start[1] = 0;
			start[2] = 0;
			end[0] = 0;
			end[1] = nRows;
			end[2] = nCols;
			break;
		case 5: //Traverse   ROW (max -> min)
			dir[0] = 1;
			dir[1] = -1;
			dir[2] = -1;
			start[0] = 0;
			start[1] = nSheets;
			start[2] = nRows;
			end[0] = nCols;
			end[1] = 0;
			end[2] = 0;
			break;
		case 6: //Traverse   ROW (min -> max)
			dir[0] = 1;
			dir[1] = -1;
			dir[2] = 1;
			start[0] = 0;
			start[1] = nSheets;
			start[2] = 0;
			end[0] = nCols;
			end[1] = 0;
			end[2] = nRows;
			break;
		default: break;
	}

	//(x,y,z Sizes are reflected the non-zero boundary in the respective index)
	int xSize = (start[0]) ? start[0] : end[0];
	int ySize = (start[1]) ? start[1] : end[1];
	int zSize = (start[2]) ? start[2] : end[2];

	//Initialize array
	unsigned char* h_voxelData;
	int size = nRows * nCols * nSheets;
	h_voxelData = new unsigned char[size];

	//Read in h_voxelData from raw file
	printf("Reading %s file...\n", fileName);
	std::ifstream rawFile(fileName);
	rawFile.read(reinterpret_cast<char*>(h_voxelData), size);
  rawFile.close();

	//Project voxel grid.
	int i = 0;
	unsigned char* h_voxel_oriented = new unsigned char[size];
	for (int x=start[0]; cmp(x, end[0]); x+=dir[0])
		for (int y=start[1]; cmp(y, end[1]); y+=dir[1])
			for (int z=start[2]; cmp(z, end[2]); z+=dir[2]) {
				h_voxel_oriented[z*xSize*ySize + y*xSize + x] = h_voxelData[i];
				i++;
			}



	//Copy voxel data to GPU.
	printf("Copying voxel data to GPU buffer...\n");
	unsigned char* d_voxelData;
	size_t voxel_bufSize = size * sizeof(unsigned char);
	hipMalloc((void**)&d_voxelData, voxel_bufSize);
	hipMemcpy(d_voxelData, h_voxel_oriented, voxel_bufSize, hipMemcpyHostToDevice);


	//Allocate image buffers on host.
	printf("Allocating Host buffers...\n");
	int projectionSize = xSize * ySize;
	unsigned char* h_maxBuf = new unsigned char[projectionSize];
	unsigned char* h_sumBuf = new unsigned char[projectionSize];
	float* h_workBuf = new float[projectionSize];

	//Allocate a location on the GPU to store the maxWeightedSum
	float* d_maxWeightedSum;
	hipMalloc((void**)&d_maxWeightedSum, sizeof(float));

	//Allocate GPU buffers for images in device memory.
	printf("Allocating image buffers on GPU...\n");
	size_t imageBufSize = projectionSize * sizeof(unsigned char);
	unsigned char* d_maxBuf;	//Stores result for MAX image.
	hipMalloc((void**)&d_maxBuf, imageBufSize);
	unsigned char* d_sumBuf; //Stores result for SUM image.
	hipMalloc((void**)&d_sumBuf, imageBufSize);
	float* d_workBuf; //Working buffer for SUM computation.
	hipMalloc((void**)&d_workBuf, projectionSize * sizeof(int));

	//Compute threads/block and blocks/grid.
	// NOTE: I did explore using the occupancy calculation to maximize throughput/
	// utilization, but struggled visualizing how to work with it and returned to this approach
	int blocksPerGrid = xSize;
	int threadsPerBlock = ySize;

	printf("\nPreparing to invoke kernels - Reviewing calculated parameters:\n");
	printf("Size: %d\n", size);
	printf("Blocks per Grid: %d\n", blocksPerGrid);
	printf("Threads per Block: %d\n", threadsPerBlock);
	printf("voxel_bufSize: %d\n", (int)voxel_bufSize);
	printf("\n");

	//Invoke MAX image Kernel.
	maxImage<<<blocksPerGrid, threadsPerBlock>>>(d_voxelData, d_maxBuf, d_workBuf, d_maxWeightedSum, zSize);
	hipDeviceSynchronize();

	//Invoke SUM image Kernel.
	sumImage<<<blocksPerGrid, threadsPerBlock>>>(d_voxelData, d_sumBuf, d_workBuf, d_maxWeightedSum, zSize);
	hipDeviceSynchronize();

	// Copy resulting MAX and SUM images from device memory to host memory
	printf("Copying results back to host...\n");
	hipMemcpy(h_maxBuf, d_maxBuf, imageBufSize, hipMemcpyDeviceToHost);
	hipMemcpy(h_sumBuf, d_sumBuf, imageBufSize, hipMemcpyDeviceToHost);

	// Free device memory
	printf("Freeing device memory...\n");
	hipFree(d_voxelData);
	hipFree(d_maxBuf);
	hipFree(d_sumBuf);
	hipFree(d_workBuf);
	hipFree(d_maxWeightedSum);

	//Write the output images.
	printf("Writing images...\n");
	char fName[(sizeof(outputFileNameBase) + 7*sizeof(char))/sizeof(char)];
	sprintf(fName, "%s%s", outputFileNameBase, "MAX.png");
	writeImage(fName, xSize, ySize, h_maxBuf);
	sprintf(fName, "%s%s", outputFileNameBase, "SUM.png");
	writeImage(fName, xSize, ySize, h_sumBuf);


	//Free up allocated memory from host.
	printf("Freeing memory from host...\n");
	delete[] h_voxelData;
	delete[] h_voxel_oriented;
	delete[] h_maxBuf;
	delete[] h_sumBuf;
	delete[] h_workBuf;
	return 0;
}
